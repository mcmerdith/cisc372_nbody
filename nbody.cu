#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include "vector.hpp"
#include "config.hpp"
#include "planets.hpp"
#include "cuda_compute.hpp"

// represents the objects in the system.  Global variables
double3 *d_hVel, *d_hPos;
vector3 *hVel, *hPos;
double *mass, *d_mass;

// initHostMemory: Create storage for numObjects entities in our system
// Parameters: numObjects: number of objects to allocate
// Returns: None
// Side Effects: Allocates memory in the hVel, hPos, and mass global variables
void initHostMemory(int numObjects)
{
	hVel = (vector3 *)malloc(sizeof(vector3) * numObjects);
	hPos = (vector3 *)malloc(sizeof(vector3) * numObjects);
	mass = (double *)malloc(sizeof(double) * numObjects);
}

// initDeviceMemory: Create storage for numObjects entities in our system
// Parameters: numObjects: number of objects to allocate
// Returns: None
// Side Effects: Allocates memory in the d_hVel, d_hPos, and d_mass global variables
void initDeviceMemory(int numObjects)
{
	hipMalloc(&d_hVel, sizeof(vector3) * numObjects);
	hipMalloc(&d_hPos, sizeof(vector3) * numObjects);
	hipMalloc(&d_mass, sizeof(double) * numObjects);
}

// freeHostMemory: Free storage allocated by a previous call to initHostMemory
// Parameters: None
// Returns: None
// Side Effects: Frees the memory allocated to global variables hVel, hPos, and mass.
void freeHostMemory()
{
	free(hVel);
	free(hPos);
	free(mass);
}

// freeDeviceMemory: Free storage allocated by a previous call to initDeviceMemory
// Parameters: None
// Returns: None
// Side Effects: Frees the memory allocated to global variables d_hVel, d_hPos, and d_mass.
void freeDeviceMemory()
{
	hipFree(d_hVel);
	hipFree(d_hPos);
	hipFree(d_mass);
}

// planetFill: Fill the first NUMPLANETS+1 entries of the entity arrays with an estimation
//				of our solar system (Sun+NUMPLANETS)
// Parameters: None
// Returns: None
// Fills the first 8 entries of our system with an estimation of the sun plus our 8 planets.
void planetFill()
{
	int i, j;
	double data[][7] = {SUN, MERCURY, VENUS, EARTH, MARS, JUPITER, SATURN, URANUS, NEPTUNE};
	for (i = 0; i <= NUMPLANETS; i++)
	{
		for (j = 0; j < 3; j++)
		{
			hPos[i][j] = data[i][j];
			hVel[i][j] = data[i][j + 3];
		}
		mass[i] = data[i][6];
	}
}

// randomFill: FIll the rest of the objects in the system randomly starting at some entry in the list
// Parameters: 	start: The index of the first open entry in our system (after planetFill).
//				count: The number of random objects to put into our system
// Returns: None
// Side Effects: Fills count entries in our system starting at index start (0 based)
void randomFill(int start, int count)
{
	int i, j, c = start;
	for (i = start; i < start + count; i++)
	{
		for (j = 0; j < 3; j++)
		{
			hVel[i][j] = (double)rand() / RAND_MAX * MAX_DISTANCE * 2 - MAX_DISTANCE;
			hPos[i][j] = (double)rand() / RAND_MAX * MAX_VELOCITY * 2 - MAX_VELOCITY;
			mass[i] = (double)rand() / RAND_MAX * MAX_MASS;
		}
	}
}

// printSystem: Prints out the entire system to the supplied file
// Parameters: 	handle: A handle to an open file with write access to prnt the data to
// Returns: 		none
// Side Effects: Modifies the file handle by writing to it.
void printSystem(FILE *handle)
{
	int i, j;
	for (i = 0; i < NUMENTITIES; i++)
	{
		fprintf(handle, "pos=(");
		for (j = 0; j < 3; j++)
		{
			fprintf(handle, "%lf,", hPos[i][j]);
		}
		printf("),v=(");
		for (j = 0; j < 3; j++)
		{
			fprintf(handle, "%lf,", hVel[i][j]);
		}
		fprintf(handle, "),m=%lf\n", mass[i]);
	}
}

int main(int argc, char **argv)
{
	clock_t t0 = clock();
	int t_now;
	// srand(time(NULL));
	srand(1234);
	initHostMemory(NUMENTITIES);
	initDeviceMemory(NUMENTITIES);
	planetFill();
	randomFill(NUMPLANETS + 1, NUMASTEROIDS);
// now we have a system.
#ifdef DEBUG
	printSystem(stdout);
#endif
	printf("We are now simulating the universe!\n");

	// send data to device
	hipMemcpy(d_hPos, hPos, NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(d_hVel, hVel, NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(d_mass, mass, NUMENTITIES, hipMemcpyHostToDevice);

	double3 **accelerations, *accel_sum;
	hipMalloc(&accelerations, sizeof(double3) * NUMENTITIES * NUMENTITIES);
	hipMalloc(&accel_sum, sizeof(double3) * NUMENTITIES);

	dim3 blockSize(32, 32);
	dim3 nBlocks((NUMENTITIES + blockSize.x - 1) / blockSize.x, (NUMENTITIES + blockSize.y - 1) / blockSize.y);

	for (t_now = 0; t_now < DURATION; t_now += INTERVAL)
	{
		compute_accelerations<<<nBlocks, blockSize>>>(accelerations, d_hPos, d_mass);
		sum_matrix<<<nBlocks, blockSize>>>(accel_sum, accelerations);
		update_positions<<<(NUMENTITIES + 1023) / 1024, 1024>>>(accel_sum, d_hVel, d_hPos);
	}

	hipMemcpy(hPos, d_hPos, NUMENTITIES, hipMemcpyDeviceToHost);
	hipMemcpy(hVel, d_hVel, NUMENTITIES, hipMemcpyDeviceToHost);
	hipMemcpy(mass, d_mass, NUMENTITIES, hipMemcpyDeviceToHost);

	clock_t t1 = clock() - t0;
#ifdef DEBUG
	printSystem(stdout);
#endif
	printf("This took a total time of %f seconds\n", (double)t1 / CLOCKS_PER_SEC);

	freeHostMemory();
	freeDeviceMemory();
}
